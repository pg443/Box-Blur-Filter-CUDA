#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void blur_filter_kernel (const float *in, float *out, int size)
{
    int i, j, k;
    int row, col;
    int curr_row, curr_col;
    float blur_value;
    int num_neighbors;

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x; 

    for (k = thread_id; k < size * size; k+= stride){
        row = k/size; 
        col = k % size; 

        blur_value = 0.0; 
        num_neighbors = 0; 
        for (i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
            for (j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
                /* Accumulate values of neighbors while checking for 
                 * boundary conditions */
                curr_row = row + i;
                curr_col = col + j;
                if ((curr_row > -1) && (curr_row < size) && (curr_col > -1) && (curr_col < size)) {
                    blur_value += in[curr_row * size + curr_col];
                    num_neighbors += 1;
                }
            }
        }

        /* Write averaged blurred value out */
        out[k] = blur_value/num_neighbors;
    }
    return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
